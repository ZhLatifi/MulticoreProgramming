#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

// Define the gpuErrchk macro
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
	{
		if (code != hipSuccess)
		{
			fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
			if (abort) exit(code);
		}
	}

__global__ void calcThreadKernel(int *block, int *warp, int *local_index) {
	int bd = blockDim.x;
	int bx = blockIdx.x;
	int tx = threadIdx.x;

	int global_idx = bd * bx + tx;
	block[global_idx] = bx;
	warp[global_idx] = tx / warpSize;
	local_index[global_idx] = tx;
}

int main(int argc, char **argv) {
	dim3 NUM_THREADS(64, 1, 1);
	dim3 NUM_BLOCKS(2, 1, 1);

	int size = NUM_THREADS.x * NUM_BLOCKS.x;
	int *block, *warp, *local_index;
	int *block_host, *warp_host, *local_index_host;

	gpuErrchk(hipMalloc((void **)&block, size * sizeof(int)));
	gpuErrchk(hipMalloc((void **)&warp, size * sizeof(int)));
	gpuErrchk(hipMalloc((void **)&local_index, size * sizeof(int)));

	calcThreadKernel << <NUM_BLOCKS, NUM_THREADS >> > (block, warp, local_index);
	gpuErrchk(hipGetLastError());

	gpuErrchk(hipDeviceSynchronize());

	block_host = (int*)malloc(size * sizeof(int));
	warp_host = (int*)malloc(size * sizeof(int));
	local_index_host = (int*)malloc(size * sizeof(int));

	gpuErrchk(hipMemcpy(block_host, block, size * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(warp_host, warp, size * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(local_index_host, local_index, size * sizeof(int), hipMemcpyDeviceToHost));

	for (int i = 0; i < size; i++) {
		printf("Calculated Thread: %d,\tBlock: %d,\tWarp: %d,\tThread: %d\n", i, block_host[i], warp_host[i], local_index_host[i]);
	}

	gpuErrchk(hipFree(block));
	gpuErrchk(hipFree(warp));
	gpuErrchk(hipFree(local_index));

	free(block_host);
	free(warp_host);
	free(local_index_host);

	return 0;
}