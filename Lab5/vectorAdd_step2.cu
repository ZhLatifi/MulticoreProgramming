#include "hip/hip_runtime.h"
/*
*	In His Exalted Name
*	Vector Addition - Sequential Code
*	Ahmad Siavashi, Email: siavashi@aut.ac.ir
*	21/05/2018
*/
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""

void fillVector(int * v, size_t n);
void addVector(int * a, int *b, int *c, size_t n);
void printVector(int * v, size_t n);
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

double start_time, elapsed_time;
__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	const int vectorSize = 1024;
	int a[vectorSize], b[vectorSize], c[vectorSize];
	
	fillVector(a, vectorSize);
	fillVector(b, vectorSize);
	
	start_time = clock();
	addWithCuda(c, a, b, vectorSize);
	//addVector(a, b, c, vectorSize);
	elapsed_time = clock() - start_time;

	// printVector(c, vectorSize);

	printf("Elapsed Time : %.10f", elapsed_time);

	return EXIT_SUCCESS;
}

// Fills a vector with data
void fillVector(int * v, size_t n) {
	int i;
	for (i = 0; i < n; i++) {
		v[i] = i;
	}
}

// Adds two vectors
void addVector(int * a, int *b, int *c, size_t n) {
	int i;
	for (i = 0; i < n; i++) {
		c[i] = a[i] + b[i];
	}
}

// Prints a vector to the stdout.
void printVector(int * v, size_t n) {
	int i;
	printf("[-] Vector elements: ");
	for (i = 0; i < n; i++) {
		printf("%d, ", v[i]);
	}
	printf("\b\b  \n");
}

// 
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size) {
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
	}

	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}

	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
	}

	addKernel << <1, 1024 >> > (dev_c, dev_a, dev_b);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}

	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
	}

	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
